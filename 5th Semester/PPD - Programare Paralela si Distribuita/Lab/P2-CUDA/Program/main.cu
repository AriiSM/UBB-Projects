#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>
#include <chrono>
#include <vector>

using namespace std;

// Variabile globale pentru dimensiuni
int original_m, original_n, conv_n;
int padding;

// Funcție pentru afișarea unei matrice
void printMatrix(int** matrix, int rows, int cols) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            cout << matrix[i][j] << " ";
        }
        cout << endl;
    }
}

// Funcție pentru salvarea unei matrice într-un fișier
void saveMatrixToFile(const string& filename, int** matrix, int rows, int cols) {
    ofstream out(filename);
    if (!out.is_open()) {
        cerr << "Error opening file: " << filename << endl;
        exit(1);
    }

    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            out << matrix[i][j] << " ";
        }
        out << endl;
    }

    out.close();
}

// Citirea datelor de intrare și aplicarea bordării (padding)
void readValuesWithPadding(int**& original, int*& padded, int*& conv) {
    ifstream in("input.txt");
    if (!in.is_open()) {
        cerr << "Error opening file: input.txt" << endl;
        exit(1);
    }

    // Citim dimensiunile matricei originale
    in >> original_m >> original_n;

    // Alocăm spațiu pentru matricea originală
    original = new int*[original_m];
    for (int i = 0; i < original_m; i++) {
        original[i] = new int[original_n];
    }

    // Citim valorile matricei originale
    for (int i = 0; i < original_m; i++) {
        for (int j = 0; j < original_n; j++) {
            in >> original[i][j];
        }
    }

    // Citim dimensiunea filtrului (kernel-ul de convoluție)
    in >> conv_n;
    padding = conv_n / 2;

    // Citim kernel-ul de convoluție
    conv = new int[conv_n * conv_n];
    for (int i = 0; i < conv_n; i++) {
        for (int j = 0; j < conv_n; j++) {
            in >> conv[i * conv_n + j];
        }
    }

    // Dimensiunile matricei cu padding
    int padded_m = original_m + 2 * padding;
    int padded_n = original_n + 2 * padding;

    // Creăm matricea cu padding
    padded = new int[padded_m * padded_n]();

    // Copiem matricea originală în centrul matricei cu padding
    for (int i = 0; i < original_m; i++) {
        for (int j = 0; j < original_n; j++) {
            padded[(i + padding) * padded_n + (j + padding)] = original[i][j];
        }
    }

    in.close();

    // Afișăm matricea originală
    cout << "Original matrix:" << endl;
    printMatrix(original, original_m, original_n);

    // Afișăm kernel-ul de convoluție
    cout << "Convolution kernel:" << endl;
    for (int i = 0; i < conv_n; i++) {
        for (int j = 0; j < conv_n; j++) {
            cout << conv[i * conv_n + j] << " ";
        }
        cout << endl;
    }
}

// Kernel CUDA pentru aplicarea convoluției
__global__ void convolutionKernel(int* padded, int* conv, int* original, int padded_m, int padded_n, int conv_n, int original_m, int original_n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y; // Rândul firului
    int col = blockIdx.x * blockDim.x + threadIdx.x; // Coloana firului

    int padding = conv_n / 2;

    if (row < original_m && col < original_n) {
        int sum = 0;

        for (int i = 0; i < conv_n; i++) {
            for (int j = 0; j < conv_n; j++) {
                int neighborRow = row + i;
                int neighborCol = col + j;
                sum += padded[neighborRow * padded_n + neighborCol] * conv[i * conv_n + j];
            }
        }

        // Salvăm rezultatul direct în matricea originală
        original[row * original_n + col] = sum;
    }
}

void compareFiles(const string& file1, const string& file2) {
    ifstream f1(file1), f2(file2);
    if (!f1.is_open()) {
        cerr << "Error opening file: " << file1 << endl;
        return;
    }
    if (!f2.is_open()) {
        cerr << "Error opening file: " << file2 << endl;
        return;
    }

    char buffer1[1024], buffer2[1024];
    int lineNumber = 0;
    bool identical = true;

    while (true) {
        // Citește o linie din fiecare fișier
        f1.getline(buffer1, sizeof(buffer1));
        f2.getline(buffer2, sizeof(buffer2));

        lineNumber++;

        // Verificăm EOF-ul pentru fiecare fișier
        bool eof1 = f1.eof();
        bool eof2 = f2.eof();

        if (eof1 && eof2) {
            break; // Ambele fișiere s-au terminat
        }
        if (eof1 || eof2) {
            cout << "Files have different lengths." << endl;
            identical = false;
            break;
        }

        // Compară liniile
        if (string(buffer1) != string(buffer2)) {
            identical = false;
            cout << "Difference at line " << lineNumber << ":\n";
            cout << "File 1: " << buffer1 << "\n";
            cout << "File 2: " << buffer2 << "\n";
        }
    }

    if (identical) {
        cout << "The files are identical." << endl;
    }

    f1.close();
    f2.close();
}


int main() {
    int **original, *padded, *conv;

    // Citirea datelor de intrare și aplicarea bordării
    readValuesWithPadding(original, padded, conv);

    // Dimensiunile matricei cu padding
    int padded_m = original_m + 2 * padding;
    int padded_n = original_n + 2 * padding;

    // Transformăm matricea 2D originală în vector 1D
    int* original_1d = new int[original_m * original_n];
    for (int i = 0; i < original_m; i++) {
        for (int j = 0; j < original_n; j++) {
            original_1d[i * original_n + j] = original[i][j];
        }
    }

    // Alocare memorie pe GPU
    int *d_padded, *d_conv, *d_original;
    hipMalloc((void**)&d_padded, padded_m * padded_n * sizeof(int));
    hipMalloc((void**)&d_conv, conv_n * conv_n * sizeof(int));
    hipMalloc((void**)&d_original, original_m * original_n * sizeof(int));

    // Copierea datelor pe GPU
    hipMemcpy(d_padded, padded, padded_m * padded_n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_conv, conv, conv_n * conv_n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_original, original_1d, original_m * original_n * sizeof(int), hipMemcpyHostToDevice);

    // Configurarea grilei și blocurilor
    dim3 threadsPerBlock(4, 4);
    dim3 numBlocks((original_n + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (original_m + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Lansarea kernel-ului CUDA
    auto start = chrono::high_resolution_clock::now();
    convolutionKernel<<<numBlocks, threadsPerBlock>>>(d_padded, d_conv, d_original, padded_m, padded_n, conv_n, original_m, original_n);
    hipDeviceSynchronize();
    auto end = chrono::high_resolution_clock::now();

    // Copierea rezultatelor înapoi în matricea originală
    hipMemcpy(original_1d, d_original, original_m * original_n * sizeof(int), hipMemcpyDeviceToHost);

    // Transformăm vectorul 1D în matrice 2D
    for (int i = 0; i < original_m; i++) {
        for (int j = 0; j < original_n; j++) {
            original[i][j] = original_1d[i * original_n + j];
        }
    }

    // Salvarea rezultatului
    saveMatrixToFile("output.txt", original, original_m, original_n);

    // Afișare timp de execuție
    cout << "Execution time: " << chrono::duration_cast<chrono::milliseconds>(end - start).count() << " ms" << endl;

    // Compararea fișierelor
    compareFiles("output.txt", "GT.txt");
    
    // Eliberare memorie
    for (int i = 0; i < original_m; i++) delete[] original[i];
    delete[] original;
    delete[] original_1d;
    delete[] padded;
    delete[] conv;
    hipFree(d_padded);
    hipFree(d_conv);
    hipFree(d_original);

    return 0;
}
